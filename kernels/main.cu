#include "hip/hip_runtime.h"
#include "main.cuh"

#include <cstdio>

#define CHK(code) \
do { \
    if ((code) != hipSuccess) { \
        fprintf(stderr, "CUDA error: %s %s %i\n", \
                        hipGetErrorString((code)), __FILE__, __LINE__); \
        exit(1); \
    } \
} while (0)

// Those functions are an example on how to call cuda functions from the main.cpp

__global__ void dev_test_vecAdd(int* A, int* B, int* C, int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= N) return;

	C[i] = A[i] + B[i];
}

void wrap_test_vectorAdd() {
	printf("Vector Add:\n");

	int N = 3;
	int a[] = { 1, 2, 3 };
	int b[] = { 1, 2, 3 };
	int c[] = { 0, 0, 0 };

	int* dev_a, * dev_b, * dev_c;

	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, N * sizeof(int));

	hipMemcpy(dev_a, a, N * sizeof(int),
		hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int),
		hipMemcpyHostToDevice);

	dev_test_vecAdd << <1, N >> > (dev_a, dev_b, dev_c, N);

	hipMemcpy(c, dev_c, N * sizeof(int),
		hipMemcpyDeviceToHost);

	hipDeviceSynchronize();

	printf("%s\n", hipGetErrorString(hipGetLastError()));

	for (int i = 0; i < N; ++i) {
		printf("%i + %i = %i\n", a[i], b[i], c[i]);
	}
}

// width, height, zi, Znear, ZFar, ZPlanes, K, R, t, inv_K, inv_R, inv_t, window, Y_cam, Y_ref, cost_cube
//__global__ void compute_cost_naive(int* width, int* height, int* zi, float* znear, float* zfar, float* ZPlanes, int* half_window,
//	std::vector<double> const& K, std::vector<double> const& R, std::vector<double> const& t, std::vector<double> const& inv_K, std::vector<double> const& inv_R, std::vector<double> const& inv_t,
//	cv::Mat const& ref, cv::Mat const& cam_1, std::vector<float*> &const cost_cube)
//{
//	int k = blockIdx.x * blockDim.x + threadIdx.x;
//	int l = blockIdx.y * blockDim.y + threadIdx.y;
//	
//}

__global__ void test_gpu(int* width, int* height, int* zi, float* znear, float* zfar, float* ZPlanes, int* half_window, double* K, double* R, double* t,
	double* inv_K, double* inv_R, double* inv_t, float* cost_cube, uint8_t* y_ref, uint8_t* y_cam)
{
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	//int l = blockIdx.y * blockDim.y + threadIdx.y;

	cost_cube[k] = (float) k;
	//if(k%100000==0) printf("GPU %d ref:%u cam:%u cost_cube:%f\n", k, y_ref[k], y_cam[k], cost_cube[k]);
	//if(k<9) printf("GPU %d KRt %f %f %f\n", k, K[k], R[k], t[k]);

}


void test(cv::Mat const& Y) {
	cv::Mat* dev_Y;
	uchar* Y_arr = Y.isContinuous()? Y.data: Y.clone().data;
	
	hipSetDevice(0);
	hipMalloc((void**)&dev_Y, sizeof(int));
	hipMemcpy(&dev_Y, Y_arr, 1920*1080*sizeof(float), hipMemcpyHostToDevice);


	cv::namedWindow("Y", cv::WindowFlags::WINDOW_AUTOSIZE);
	cv::imshow("Y", Y);
	cv::waitKey(0);
}

//void frame2frame_matching(cam& ref, cam& cam_1, std::vector<cv::Mat> &cost_cube, int zi, int half_window)
float* frame2frame_matching(cam &ref, cam &cam_1, cv::Mat &cost_cube_plane, int zi, int half_window)
{
	printf("Naive cost frame2frame_matching:\n");

	uint mat_length;
	cv::Mat mat; //, result;

	/*// Full cost cube
	mat_length = cost_cube[0].total() * cost_cube[0].channels();
	uint im_length = mat_length * 3 / 2;
	printf("size: %d\n",mat_length);

	float* new_cost_cube = new float[mat_length *ZPlanes];
	for (int i = 0; i < ZPlanes; i++)
	{
		mat = cost_cube[i];
		float* mat_arr = mat.isContinuous() ? (float*)mat.data : (float*)mat.clone().data;
		memcpy((void*) &(new_cost_cube[i* mat_length]),(void*) mat_arr, mat_length * sizeof(float));
	}*/

	// Only one plane
	mat_length = cost_cube_plane.total() * cost_cube_plane.channels();
	uint im_length = mat_length * 3 / 2;
	printf("size: %d\n", mat_length);

	float* new_cost_cube = new float[mat_length];
	float* mat_arr_plane = cost_cube_plane.isContinuous() ? (float*)cost_cube_plane.data : (float*)cost_cube_plane.clone().data;
	memcpy((void*)new_cost_cube, (void*)mat_arr_plane, mat_length * sizeof(float));

	mat = ref.YUV[0];
	uint8_t* y_ref = new uint8_t[im_length];
	uint8_t* mat_arr = mat.isContinuous() ? (uint8_t*)mat.data : (uint8_t*)mat.clone().data;
	printf("mat size th: %d\n", mat.total()* mat.channels());
	printf("im_length: %d\n", im_length);
	memcpy((void*)y_ref, (void*)mat_arr, im_length * sizeof(uint8_t));

	printf("last val = %f\n", y_ref[im_length-1]);

	mat = cam_1.YUV[0];
	uint8_t* y_cam = new uint8_t[im_length];
	mat_arr = mat.isContinuous() ? (uint8_t*)mat.data : (uint8_t*)mat.clone().data;
	memcpy((void*)y_cam, (void*)mat_arr, im_length * sizeof(uint8_t));

	double* K = &cam_1.p.K[0]; double* R = &cam_1.p.R[0]; double* t = &cam_1.p.t[0];
	double* inv_K = &ref.p.K[0]; double* inv_R = &ref.p.R[0]; double* inv_t = &ref.p.t[0];

	int* dev_width; int* dev_height; int* dev_zi; int* dev_half_window;
	float* dev_znear; float* dev_zfar; float* dev_zplanes; float* dev_cost_cube;
	double* dev_K; double* dev_R; double* dev_t; double* dev_inv_K; double* dev_inv_R; double* dev_inv_t;
	uint8_t* dev_Y_ref; uint8_t* dev_Y_cam;

	CHK(hipSetDevice(0));

	CHK(hipMalloc((void**)&dev_width, sizeof(int)));
	CHK(hipMalloc((void**)&dev_height, sizeof(int)));
	CHK(hipMalloc((void**)&dev_zi, sizeof(int)));
	CHK(hipMalloc((void**)&dev_znear, sizeof(float)));
	CHK(hipMalloc((void**)&dev_zfar, sizeof(float)));
	CHK(hipMalloc((void**)&dev_zplanes, sizeof(float)));
	CHK(hipMalloc((void**)&dev_half_window, sizeof(int)));
	CHK(hipMalloc((void**)&dev_K, 9 * sizeof(double)));
	CHK(hipMalloc((void**)&dev_R, 9 * sizeof(double)));
	CHK(hipMalloc((void**)&dev_t, 3 * sizeof(double)));
	CHK(hipMalloc((void**)&dev_inv_K, 9 * sizeof(double)));
	CHK(hipMalloc((void**)&dev_inv_R, 9 * sizeof(double)));
	CHK(hipMalloc((void**)&dev_inv_t, 3 * sizeof(double)));
	CHK(hipMalloc((void**)&dev_Y_ref, im_length * sizeof(uint8_t)));
	CHK(hipMalloc((void**)&dev_Y_cam, im_length * sizeof(uint8_t)));
	//CHK(hipMalloc((void**)&dev_cost_cube, ZPlanes * mat_length * sizeof(float)));
	CHK(hipMalloc((void**)&dev_cost_cube, mat_length * sizeof(float)));

	CHK(hipMemcpy(dev_width, &ref.width, sizeof(int), hipMemcpyHostToDevice));
	CHK(hipMemcpy(dev_height, &ref.height, sizeof(int), hipMemcpyHostToDevice));
	CHK(hipMemcpy(dev_zi, &zi, sizeof(int), hipMemcpyHostToDevice));
	CHK(hipMemcpy(dev_znear, &ZNear, sizeof(float), hipMemcpyHostToDevice));
	CHK(hipMemcpy(dev_zfar, &ZFar, sizeof(float), hipMemcpyHostToDevice));
	CHK(hipMemcpy(dev_zplanes, &ZPlanes, sizeof(float), hipMemcpyHostToDevice));
	CHK(hipMemcpy(dev_half_window, &half_window, sizeof(int), hipMemcpyHostToDevice));
	CHK(hipMemcpy(dev_K, K, 9 * sizeof(double), hipMemcpyHostToDevice));
	CHK(hipMemcpy(dev_R, R, 9 * sizeof(double), hipMemcpyHostToDevice));
	CHK(hipMemcpy(dev_t, t, 3 * sizeof(double), hipMemcpyHostToDevice));
	CHK(hipMemcpy(dev_inv_K, inv_K, 9 * sizeof(double), hipMemcpyHostToDevice));
	CHK(hipMemcpy(dev_inv_R, inv_R, 9 * sizeof(double), hipMemcpyHostToDevice));
	CHK(hipMemcpy(dev_inv_t, inv_t, 3 * sizeof(double), hipMemcpyHostToDevice));
	CHK(hipMemcpy(dev_Y_ref, y_ref, im_length * sizeof(uint8_t), hipMemcpyHostToDevice));
	CHK(hipMemcpy(dev_Y_cam, y_cam, im_length * sizeof(uint8_t), hipMemcpyHostToDevice));
	//CHK(hipMemcpy(dev_cost_cube, new_cost_cube, ZPlanes * mat_length * sizeof(float), hipMemcpyHostToDevice));

	int N_threads = 1024;
	dim3 thread_size(N_threads);
	dim3 block_size((mat_length + N_threads - 1) / N_threads);

	/*compute_cost_naive << <block_size, thread_size>> > (dev_width, dev_height, dev_zi, dev_znear, dev_zfar, dev_zplanes, dev_half_window, dev_cam_K,
		dev_cam_R, dev_cam_t, dev_ref_inv_K, dev_ref_inv_R, dev_ref_inv_t, dev_Y_ref, dev_Y_cam, dev_cost_cube);*/

	test_gpu << <block_size, thread_size >> > (dev_width, dev_height, dev_zi, dev_znear, dev_zfar, dev_zplanes, dev_half_window, dev_K, dev_R, dev_t,
		dev_inv_K, dev_inv_R, dev_inv_t, dev_cost_cube, dev_Y_ref, dev_Y_cam);
	
	//CHK(hipGetLastError());
	hipGetLastError();

	printf("Avant print CPU");

	for(int k=0; k<mat_length; k+=100000)  printf("CPU0 new_cost_cube %d ref:%u cam:%u cost_cube:%f\n", k, y_ref[k], y_cam[k], new_cost_cube[k]);
	//for (int k = 0; k < 9; k++) printf("CPU %d KRt %f %f %f\n", k, K[k], R[k], t[k]);
	
	printf("Thread size %d, block size %d\n", thread_size.x, block_size.x);

	//CHK(hipMemcpy(cost_cube_plane.data, dev_cost_cube, mat_length * sizeof(float), hipMemcpyDeviceToHost));
	CHK(hipMemcpy(new_cost_cube, dev_cost_cube, mat_length * sizeof(float), hipMemcpyDeviceToHost));
	//memcpy((void*)result.data, (void*)&new_cost_cube[0], mat_length * sizeof(float));
	//cv::Mat result = cv::Mat(ref.width, ref.height, CV_32FC1, &new_cost_cube);
	float* result = new_cost_cube;

	for (int k = 0; k < mat_length; k += 100000)  printf("CPU1 new_cost_cube %d ref:%u cam:%u cost_cube:%f\n", k, y_ref[k], y_cam[k], new_cost_cube[k]);
	CHK(hipFree(dev_width));
	CHK(hipDeviceReset());
	return result;
}